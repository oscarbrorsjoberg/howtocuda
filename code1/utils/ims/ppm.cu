#include "hip/hip_runtime.h"
#include <assert.h>
#include <getopt.h>
#include <cstring>
#include <fstream>
#include <iostream>
#include <memory>
#include <hip/hip_runtime_api.h>

#include "utils.h"
#include "ims/ims.hpp"

/******************************************************************************
* File:             ppm.cu
*
* Author:             
* Created:          04/14/22 
* Description:      A ppm read write for cuda alignment
*****************************************************************************/

__global__ void unpack_image(planar_image_t planar, const pixel_t *packed, int pixel_count)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= pixel_count) return;

  planar.r[index] = packed[index].r;
  planar.g[index] = packed[index].g;
  planar.b[index] = packed[index].b;
}

__global__ void pack_image(const planar_image_t planar, pixel_t *packed, int pixel_count)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= pixel_count) return;

  packed[index].r = planar.r[index];
  packed[index].g = planar.g[index];
  packed[index].b = planar.b[index];
}


static const unsigned int HEADER_SIZE = 0x40;
static const unsigned int CHANNELS = 3;

/******************************************************************************
* Function:         
* Description:      
* Where:
* Return:           
* Error:            
*****************************************************************************/

static bool loadPPM(const char *file, pixel_t **data, unsigned int *w, unsigned int *h)
{
  FILE *fp = fopen(file, "rb");

  if (!fp) {
    std::cerr << "loadPPM() : failed to open file: " << file << "\n";
    return false;
  }

  // check header
  char header[HEADER_SIZE];

  if (fgets(header, HEADER_SIZE, fp) == nullptr) {
    std::cerr << "loadPPM(): reading header returned NULL\n";
    return false;
  }

  if (strncmp(header, "P6", 2)) {
    std::cerr << "unsupported image format\n";
    return false;
  }

  // parse header, read maxval, width and height
  unsigned int width = 0;
  unsigned int height = 0;
  unsigned int maxval = 0;
  unsigned int i = 0;

  while (i < 3) {
    if (fgets(header, HEADER_SIZE, fp) == NULL) {
      std::cerr << "loadPPM() : reading PPM header returned NULL" << std::endl;
      return false;
    }

    if (header[0] == '#') {
      continue;
    }

    if (i == 0) {
      i += sscanf(header, "%u %u %u", &width, &height, &maxval);
    } else if (i == 1) {
      i += sscanf(header, "%u %u", &height, &maxval);
    } else if (i == 2) {
      i += sscanf(header, "%u", &maxval);
    }
  }

  size_t pixel_count = width * height;
  size_t data_size = sizeof(unsigned char) * pixel_count * CHANNELS;
  unsigned char *raw_data = static_cast<unsigned char *>(malloc(data_size));
  *w = width;
  *h = height;

  // read and close file
  if (fread(raw_data, sizeof(unsigned char), pixel_count * CHANNELS, fp) == 0) {
    std::cerr << "loadPPM() read data returned error.\n";
  }
  fclose(fp);

  pixel_t *pixel_data = static_cast<pixel_t*>(malloc(pixel_count * sizeof(pixel_t)));
  float scale = 1.0f / 255.0f;
  for (int i = 0; i < pixel_count; i++) {
    pixel_data[i].r = raw_data[3 * i + 0] * scale;
    pixel_data[i].g = raw_data[3 * i + 1] * scale;
    pixel_data[i].b = raw_data[3 * i + 2] * scale;
  }

  *data = pixel_data;
  free(raw_data);

  return true;
}
/******************************************************************************
* Function:         
* Description:      
* Where:
* Return:           
* Error:            
*****************************************************************************/
static bool savePPM(const std::string &file, const pixel_t *pixels, 
                          int width, int height)
{
  assert(pixels != nullptr);
  assert(width > 0);
  assert(height > 0);

  std::fstream fh(file, std::fstream::out | std::fstream::binary);

  if (fh.bad()) {
    std::cerr << "savePPM() : open failed.\n";
    return false;
  }

  fh << "P6\n";
  fh << width << "\n" << height << "\n" << 0xff << "\n";

  unsigned int pixel_count = width * height;

  for (unsigned int i = 0; (i < pixel_count) && fh.good(); ++i) {
    fh << static_cast<unsigned char>(pixels[i].r* 255);
    fh << static_cast<unsigned char>(pixels[i].g * 255);
    fh << static_cast<unsigned char>(pixels[i].b * 255);
  }

  fh.flush();

  if (fh.bad()) {
    std::cerr << "savePPM() : writing data failed.\n";
    return false;
  }

  fh.close();
  return true;
}

planar_image_t planar_image_create(int width, int height)
{
  planar_image_t out;

  out.width = width;
  out.height = height;

 int pixel_count = width * height;

 ck(hipMalloc(&out.r, pixel_count * sizeof(float)));
 ck(hipMalloc(&out.g, pixel_count * sizeof(float)));
 ck(hipMalloc(&out.b, pixel_count * sizeof(float)));

  return out;
}

void planar_image_free(planar_image_t &img)
{
  ck(hipFree(img.r));
  ck(hipFree(img.g));
  ck(hipFree(img.b));
}

// this is stolen from corse, why 128?
constexpr int BLOCK_SIZE = 128;

/*
   reads host ppm and creates planar image on device from host
   */


bool CU_readppm_planar_image(
    const std::string &input_path, planar_image_t &device_image)
{
	unsigned int width, height;
  // loading pixels (as (rgb per pixel))
	pixel_t *host_pixels = nullptr;
	if (!loadPPM(input_path.c_str(),
				&host_pixels, &width, &height)) {
		std::cerr << "Couldn't read image " << input_path << "\n";
    return false;
	}
  if(host_pixels){
    int pixel_count = width * height;

    device_image = planar_image_create(width, height);
    size_t image_size = pixel_count * sizeof(pixel_t);
    pixel_t *device_pixels;

    ck(hipMalloc(&device_pixels, (int)image_size));
    ck(hipMemcpy(device_pixels, host_pixels, (int)image_size, hipMemcpyHostToDevice));

    // number of pixels per block?
    int number_blocks = (pixel_count + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // unpack image create planar image
    unpack_image<<<number_blocks, BLOCK_SIZE>>>(device_image, device_pixels,
        pixel_count);

    ck(hipFree(device_pixels));
    free(host_pixels);
  }
  return true;
}

/*
  --   saves image as ppm (by packing it to pixels)
*/

bool CU_saveppm_planar_image(
    const std::string &output_path, const planar_image_t &device_image)
{

  int pixel_count = device_image.width * device_image.height;
  int number_blocks = (pixel_count + BLOCK_SIZE - 1) / BLOCK_SIZE;

  pixel_t *host_pixels;
  pixel_t *dev_pixels;

  int image_size = pixel_count * (int)sizeof(pixel_t);

  ck(hipMalloc(&dev_pixels, image_size));

  host_pixels = new pixel_t[pixel_count];

  // unpack image create planar image
  pack_image<<<number_blocks, BLOCK_SIZE>>>(device_image, dev_pixels,
                                              pixel_count);
  ck(hipMemcpy(host_pixels, dev_pixels, image_size, hipMemcpyDeviceToHost));

  ck(hipFree(dev_pixels));

  if(!savePPM(output_path, host_pixels, device_image.width, device_image.height)){
    std::cerr << "Unable to save image " << output_path << "\n";
    return false;
  }

  delete[] host_pixels;

  return true;
}

