
// Utility functions for example programs.

#include <assert.h>
#include <getopt.h>
#include <cstring>
#include <fstream>
#include <iostream>
#include <memory>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

#include "utils.h"



KernelTimer::KernelTimer()
{
	hipEventCreate(&start_);
	hipEventCreate(&stop_);
}

void KernelTimer::start(){
	hipEventRecord(start_);
}

void KernelTimer::stop(){
	hipEventRecord(stop_);
}

KernelTimer::~KernelTimer()
{
	float ms = 0.0;
	hipEventSynchronize(stop_);
	hipEventElapsedTime(&ms, start_, stop_);

	std::cout << "Kernel ran in " << ms << "ms \n";
	/* hipEventDestroy(&start_); */
	/* hipEventDestroy(&stop_); */
}

void printCudaInformation()
{

	int darabszam;
	int driverVersion = 0, runtimeVersion = 0;
	int devCount;


	hipGetDeviceCount(&devCount);
	if(devCount == 0){
		std::cout << "No supported CUDA device found\n";
	}
	else{
		std::cout << devCount<< "CUDA device(s) found\n";
	}

	for(int dev = 0; dev < devCount; ++dev){
		hipSetDevice(dev);
		hipDeviceProp_t deviceProp;
		cudeGetDeviceProperties(&deviceProp, dev);

		std::cout << dev << "device name" 
			<< deviceProp.name << "\n";

		std::cout << "CUDA capability version" <<
			<< deviceProp.major << " " << deviceProp.minor << "\n";

		hipDriverGetVersion(&driveVersion);
		hipRuntimeGetVersion(&runtimeVersion);

		std::cout << "CUDA driver verison / Runtime version " <<
			driverVersion / 1000 << " " << (driverVersion % 100) / 10
			<< "/" <<
			runtimeVersion / 1000 << " " << (driverVersion % 100) / 10 <<
	}

}


