
// Utility functions for example programs.

#include <assert.h>
#include <getopt.h>
#include <cstring>
#include <fstream>
#include <iostream>
#include <memory>
#include <hip/hip_runtime_api.h>

#include "utils.h"



KernelTimer::KernelTimer()
{
	hipEventCreate(&start_);
	hipEventCreate(&stop_);
}

void KernelTimer::start(){
	hipEventRecord(start_);
}

void KernelTimer::stop(){
	hipEventRecord(stop_);
}

KernelTimer::~KernelTimer()
{
	float ms = 0.0;
	hipEventSynchronize(stop_);
	hipEventElapsedTime(&ms, start_, stop_);

	std::cout << "Kernel ran in " << ms << "ms \n";
	/* hipEventDestroy(&start_); */
	/* hipEventDestroy(&stop_); */
}
