#include <iostream>
#include <hip/hip_runtime_api.h>

#include "KernelTimer.hpp"

KernelTimer::KernelTimer()
{
	hipEventCreate(&start_);
	hipEventCreate(&stop_);
}

void KernelTimer::start(){
	hipEventRecord(start_);
}

void KernelTimer::stop(){
	hipEventRecord(stop_);
}

KernelTimer::~KernelTimer()
{
	float ms = 0.0;
	hipEventSynchronize(stop_);
	hipEventElapsedTime(&ms, start_, stop_);

	std::cout << "Kernel ran in " << ms << "ms \n";
}
