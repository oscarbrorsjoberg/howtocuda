#include <assert.h>
#include <getopt.h>
#include <cstring>
#include <fstream>
#include <iostream>
#include <memory>
#include <hip/hip_runtime_api.h>

/* #include <hip/hip_runtime_api.h> */
#include "KernelTimer.hpp"

KernelTimer::KernelTimer()
{
	hipEventCreate(&start_);
	hipEventCreate(&stop_);
}

void KernelTimer::start(){
	hipEventRecord(start_);
}

void KernelTimer::stop(){
	hipEventRecord(stop_);
}

KernelTimer::~KernelTimer()
{
	float ms = 0.0;
	hipEventSynchronize(stop_);
	hipEventElapsedTime(&ms, start_, stop_);

	std::cout << "Kernel ran in " << ms << "ms \n";
}
